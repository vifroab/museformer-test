#include "hip/hip_runtime.h"
#include <torch/torch.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <THC/THCAtomics.cuh>


const long THREADS_PER_BLOCK = 1024;
const long MAX_GRID_NUM = 2147483647;


inline long GET_BLOCKS(const long N) {
  long optimal_block_num = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  return long(min(optimal_block_num, MAX_GRID_NUM));
}


template <typename scalar_t>
__global__ void cudaForward(
    const long * ranges,
    const scalar_t * values,
    const long input_size,
    scalar_t * output) {

    const long index = long(blockIdx.x) * long(blockDim.x) + long(threadIdx.x);

    if (index >= input_size) return;

    const long line_start = index * 2;
    const long begin_idx = ranges[line_start];
    const long end_idx = ranges[line_start + 1];

    long value = values[index];
    for (long idx = begin_idx; idx < end_idx; idx++) {
        output[idx] = value;
    }
}


int cudaForwardLauncher(
    const at::Tensor& ranges,
    const at::Tensor& values,
    const long num_chunks,
    at::Tensor& output
) {
    const long input_size = num_chunks;
    assert (input_size <= THREADS_PER_BLOCK * MAX_GRID_NUM);

    AT_DISPATCH_INTEGRAL_TYPES(
        values.type(), "cudaForward",
        ([&] {
            const long *ranges_ = ranges.data_ptr<long>();
            const scalar_t *values_ = values.data_ptr<scalar_t>();
            scalar_t *output_ = output.data_ptr<scalar_t>();

            cudaForward<<<GET_BLOCKS(input_size), THREADS_PER_BLOCK>>>(
                ranges_, values_, input_size, output_
            );
          }
        )
    );

    THCudaCheck(hipGetLastError());

    return 0;
}
